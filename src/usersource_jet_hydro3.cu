#include "hip/hip_runtime.h"


__device__ __host__
int addsourceterms2_MODID(real *dw, real *wd, real *w, struct params *p, struct state *s,int *ii,int field,int dir) {

  int direction;
  int status=0;

   real xc1,xc2,r1,r2;
   real xxmax,yymax;
   real dx,dy,dz;
   real aa;
   real s_period;
   real qt, tdep;
   real s_rad1,s_rad2;
   real exp_z, exp_xy;

   real vvv;



   real xp,yp,zp;
   int i,j,k;
   int n1;
 	  
	  i=ii[0];
	  j=ii[1];

     xc1=4.0e6;
    xc2=600000.0;
    qt=p->qt;

    aa=10;
    n1=1;

          xp=(p->xmin[0])+(((real)i)*(p->dx[0]));
          yp=(p->xmin[1])+(((real)j)*(p->dx[1]));
     
   xxmax=(p->xmax[0])-2*((p->dx[0]));
    s_period=300.0;
    //tdep=1.00;
    tdep=sin(qt*2.0*PI/s_period);

     r1=(xp-xc1)*(xp-xc1);
     r2=(yp-xc2)*(yp-xc2);

 
 s_rad1=1000.0;
 s_rad2=10000.0;
 exp_z=exp(-(r2/(s_rad2*s_rad2)));
 exp_xy=sin(PI*xp*(n1+1)/xxmax)*exp_z;
      vvv=aa*tdep*exp_xy;

     // if(i==3 && j==149)
      //if(i==3 && j==200)
     /* {
                  p->test=vv;
               p->chyp[0]=xp;
                p->chyp[1]=yp;
       }*/

       /* if(i==9 && j==63 && k==63) 
	{
                p->test=(w[fencode3_MODID(p,ii,rho)]);
                p->chyp[0]=vx;
                p->chyp[1]=vy;
                p->chyp[2]=(w[fencode3_MODID(p,ii,mom1)]);
	}*/


//if(i==512 && j==15 )
//printf("%g %g  %g %g %g \n", xp, yp,vvv,r1, r2);
 

 
                           w[fencode3_MODID(p,ii,mom2)]=w[fencode3_MODID(p,ii,mom2)]+(p->dt)*vvv*(w[fencode3_MODID(p,ii,rho)]+w[fencode3_MODID(p,ii,rhob)]);
  
                          w[fencode3_MODID(p,ii,energy)]=w[fencode3_MODID(p,ii,energy)]+(p->dt)*(vvv*vvv)*(w[fencode3_MODID(p,ii,rho)]+w[fencode3_MODID(p,ii,rhob)])/2.0;
  

  return ( status);
}

__device__ __host__
int addsourceterms1_MODID(real *dw, real *wd, real *w, struct params *p, struct state *s,int *ii,int field,int dir) {

   

}


#include "hip/hip_runtime.h"


__device__ __host__
int addsourceterms2_MODID(real *dw, real *wd, real *w, struct params *p, struct state *s,int *ii,int field,int dir) {

  int direction;
  int status=0;

   real xc1,xc2,r1,r2;
   real xxmax,yymax;
   real dx,dy,dz;
   real aa;
   real s_period;
   real qt, tdep;
   real s_rad1,s_rad2;
   real exp_z, exp_x, exp_xy, exp_y;

   real vvv;

   real tim, timtanh;



   real xp,yp,zp;
   int i,j,k;
   int n1;
 	  
	  i=ii[0];
	  j=ii[1];

     xc1=4.0e6;
    xc2=300000.0;
    qt=p->qt;

    //if(qt<300)
    //   xc2=xc2+125000;

    aa=1000;
    n1=1;

          xp=(p->xmin[0])+(((real)i)*(p->dx[0]));
          yp=(p->xmin[1])+(((real)j)*(p->dx[1]));
     
   xxmax=(p->xmax[0])-2*((p->dx[0]));
    s_period=100.0;
    tdep=exp(-(qt-s_period)*(qt-s_period));
    //tdep=sin(qt*2.0*PI/s_period);

     r1=(xp-xc1)*(xp-xc1);
     r2=(yp-xc2)*(yp-xc2);

 
 s_rad1=1870.0; // from Mackenzie dover app 913:19(10pp), 2021 May 20
 s_rad2=1000000.0;
 exp_x=exp(-(r1/(s_rad1*s_rad1)));
 exp_y=exp(-(r2/(s_rad2*s_rad2)));
 //exp_xy=sin(PI*xp*(n1+1)/xxmax)*exp_z;

   tim=PI+(PI*((qt-s_period)/s_period)); 
   timtanh=(exp(2*tim)-1)/(exp(2*tim)+1);
   exp_xy=-(1+timtanh)*exp_x*exp_y/2;
   

      vvv=aa*tdep*exp_xy;

     // if(i==3 && j==149)
      //if(i==3 && j==200)
     /* {
                  p->test=vv;
               p->chyp[0]=xp;
                p->chyp[1]=yp;
       }*/

       /* if(i==9 && j==63 && k==63) 
	{
                p->test=(w[fencode3_MODID(p,ii,rho)]);
                p->chyp[0]=vx;
                p->chyp[1]=vy;
                p->chyp[2]=(w[fencode3_MODID(p,ii,mom1)]);
	}*/


//if(i==512 && j==15 )
//printf("%g %g  %g %g %g \n", xp, yp,vvv,r1, r2);
 

// if(j==2 || j==3)
//{
                           w[fencode3_MODID(p,ii,mom2)]=w[fencode3_MODID(p,ii,mom2)]+(p->dt)*vvv*(w[fencode3_MODID(p,ii,rho)]+w[fencode3_MODID(p,ii,rhob)]);
  
                          w[fencode3_MODID(p,ii,energy)]=w[fencode3_MODID(p,ii,energy)]+(p->dt)*(vvv*vvv)*(w[fencode3_MODID(p,ii,rho)]+w[fencode3_MODID(p,ii,rhob)])/2.0;
//}
  

  return ( status);
}

__device__ __host__
int addsourceterms1_MODID(real *dw, real *wd, real *w, struct params *p, struct state *s,int *ii,int field,int dir) {

   

}

